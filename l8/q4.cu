#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
__global__ void add(char * d_res , char * d_buffer , int * d_length)
{
    int threadid = threadIdx.x;
    int length = *d_length ;
    d_res[threadid]=d_buffer[length-1-threadid];
    
 }
int main(void)
{

char buffer[100] ="my name is yash";
char res[100];
int length = strlen(buffer) ;

char * d_res ; 
char * d_buffer ; 
int * d_length;

hipMalloc((void **)&d_res,(length+1) * sizeof(char));
hipMalloc((void **)&d_buffer,(length+1)*sizeof(char));
 hipMalloc((void **)&d_length,sizeof(int));

hipMemcpy(d_buffer,buffer,(length+1)*sizeof(char),hipMemcpyHostToDevice);
 hipMemcpy(d_length,&length,sizeof(int),hipMemcpyHostToDevice);


add<<<1,length>>>(d_res,d_buffer,d_length);
hipError_t error = hipGetLastError();
 if(error!= hipSuccess)
 {
	 	printf("%s\n",hipGetErrorString(error));
 }
hipMemcpy(&res,d_res,sizeof(char) * length,hipMemcpyDeviceToHost);
res[length]='\0';
printf("Result :%s",res);

}