#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
__global__ void add(char * d_searchfor , char * d_buffer, int * d_start,int * d_end,int * res)
{
    int threadid = threadIdx.x;
    char temp[100];
    int pos = 0 ;
    int check = 0 ;
    for(int i = d_start[threadid];i<=d_end[threadid];i++)
    {
        if(d_searchfor[pos]=='\0')
        {
            check=1 ;
            break ;
        }
        if(d_buffer[i]!=d_searchfor[pos])
        {
            check = 1 ;
            break ;
        }
        pos ++ ;
    }
    if(check==0)
    {
        atomicAdd(res,1);
    }
 }
int main(void)
{

char buffer[100] ="a bcdef a a a xyt a";
 char searchfor[100]="a";
int wstart[50];
int wend[50];
int wc = 0 ; 
int length = strlen(buffer) ;
int slen = strlen(searchfor);
 int result = 0 ;

 wstart[0]=0 ;
 int spos = 1 ;
 int epos = 0 ;
for(int i = 0 ;i<=length-1;i++)
{
    if(buffer[i]==' ')
    {
        wc++;
        wstart[spos]=i+1 ;
        spos ++ ;
        wend[epos]=i-1 ;
        epos ++ ;
    }
}
wc++;
wend[epos]=length-1;
 
char * d_searchfor ; 
char * d_buffer ; 
int * d_start ;
int * d_end ;
int * d_res ;

hipMalloc((void **)&d_searchfor,(slen+1) * sizeof(char));
hipMalloc((void **)&d_buffer,(length+1)*sizeof(char));
hipMalloc((void **)&d_start , sizeof(int)*wc);
hipMalloc((void **)&d_end ,sizeof(int) * wc);
hipMalloc((void **)&d_res ,sizeof(int));
  
hipMemcpy(d_buffer,buffer,(length+1)*sizeof(char),hipMemcpyHostToDevice);
hipMemcpy(d_searchfor,searchfor,(slen+1)*sizeof(char),hipMemcpyHostToDevice);
hipMemcpy(d_start , wstart , sizeof(int)*wc,hipMemcpyHostToDevice);
hipMemcpy(d_end , wend , sizeof(int)*wc,hipMemcpyHostToDevice);
 hipMemcpy(d_res , &result , sizeof(int),hipMemcpyHostToDevice);


add<<<1,wc>>>(d_searchfor,d_buffer,d_start,d_end,d_res);
hipError_t error = hipGetLastError();
 if(error!= hipSuccess)
 {
	 	printf("%s\n",hipGetErrorString(error));
 }
hipMemcpy(&result,d_res,sizeof(int),hipMemcpyDeviceToHost);

printf("Result :%d",result);

}