#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
__global__ void add(char * d_resbuffer , char * d_buffer, int * d_start,int * d_end)
{
 
  int threadid = threadIdx.x ;
  int start = d_start[threadid];
  int end = d_end[threadid];
 
  int pos = start ;
 int till = end-start+1;
  for(int i=1 ;i<=till;i++)
  {
      
      
      d_resbuffer[pos]=d_buffer[end];
      pos ++ ;
      end -- ;
  }
  d_resbuffer[pos]=' ';
  
}
int main(void)
{

char buffer[100] ="what a beautiful day";
int wstart[50];
int wend[50];
int wc = 0 ; 
char res[100] = "";

int length = strlen(buffer) ;
 
 wstart[0]=0 ;
 int spos = 1 ;
 int epos = 0 ;
for(int i = 0 ;i<=length-1;i++)
{
    if(buffer[i]==' ')
    {
        wc++;
        wstart[spos]=i+1 ;
        spos ++ ;
        wend[epos]=i-1 ;
        epos ++ ;
    }
}
wc++;
wend[epos]=length-1;
 
char * d_resbuffer ; 
char * d_buffer ; 
int * d_start ;
int * d_end ;

hipMalloc((void **)&d_resbuffer,(length+1) * sizeof(char));
hipMalloc((void **)&d_buffer,(length+1)*sizeof(char));
hipMalloc((void **)&d_start , sizeof(int)*wc);
hipMalloc((void **)&d_end ,sizeof(int) * wc);
  
hipMemcpy(d_buffer,buffer,(length+1)*sizeof(char),hipMemcpyHostToDevice);
hipMemcpy(d_start , wstart , sizeof(int)*wc,hipMemcpyHostToDevice);
hipMemcpy(d_end , wend , sizeof(int)*wc,hipMemcpyHostToDevice);


add<<<1,wc>>>(d_resbuffer,d_buffer,d_start,d_end);
hipError_t error = hipGetLastError();
 if(error!= hipSuccess)
 {
	 	printf("%s\n",hipGetErrorString(error));
 }
hipMemcpy(res,d_resbuffer,(length)*sizeof(char),hipMemcpyDeviceToHost);
res[length]='\0';
 
printf("Result :%s",res);

}