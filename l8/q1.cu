#include<hip/hip_runtime.h>

#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void add(char * d_resbuffer , char * d_buffer, int * d_length)
{
		
	int id = threadIdx.x ;
	int start = id * (*d_length);
	for(int i = 0 ; i<=(*d_length)-1;i++)
	{
		d_resbuffer[start] = d_buffer[i];
		start ++ ;
	}
}
int main(void)
{

char buffer[100] ="Hello";

char res[100] = "";

int length = strlen(buffer) ;

int duplicate = 4 ;
 
char * d_resbuffer ; 
  
char * d_buffer ; 
int * d_length ;

hipMalloc((void **)&d_resbuffer,(length*duplicate+1) * sizeof(char));
hipMalloc((void **)&d_buffer,(length+1)*sizeof(char));
hipMalloc((void **)&d_length , sizeof(int));
  
hipMemcpy(d_buffer,buffer,(length+1)*sizeof(char),hipMemcpyHostToDevice);
hipMemcpy(d_length , &length , sizeof(int),hipMemcpyHostToDevice);


add<<<1,duplicate>>>(d_resbuffer,d_buffer,d_length);
hipError_t error = hipGetLastError();
 if(error!= hipSuccess)
 {
	 	printf("%s\n",hipGetErrorString(error));
 }
hipMemcpy(res,d_resbuffer,(length*duplicate)*sizeof(char),hipMemcpyDeviceToHost);

 int location = length * duplicate ;
 res[location]='\0';
 
printf("Result :%s",res);

}