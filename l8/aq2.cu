#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
__global__ void add(char * d_res , char * d_buffer)
{
    int tid = threadIdx.x;
    int start = (tid*tid + tid)/2 ;
    for(int i = 1 ;i<=tid+1;i++)
    {
        d_res[start]=d_buffer[tid];
        start++;
    }
    
 }
int main(void)
{

char buffer[100] ="abcdef";
char res[100]="";
int length = strlen(buffer) ;
int len2 = (length*(length+1))/2 + 1 ;
char * d_res ; 
char * d_buffer ; 


hipMalloc((void **)&d_res,(len2) * sizeof(char));
hipMalloc((void **)&d_buffer,(length+1)*sizeof(char));


hipMemcpy(d_buffer,buffer,(length+1)*sizeof(char),hipMemcpyHostToDevice);

add<<<1,length>>>(d_res,d_buffer);
hipError_t error = hipGetLastError();
 if(error!= hipSuccess)
 {
	 	printf("%s\n",hipGetErrorString(error));
 }
hipMemcpy(&res,d_res,sizeof(char) * (len2-1),hipMemcpyDeviceToHost);
res[len2-1]='\0';
printf("Result :%s",res);

}