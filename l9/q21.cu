#include<hip/hip_runtime.h>

#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void add(int* d_a,int* d_b,int* d_r,int *d_m)
{
	int n = threadIdx.x;
	for(int i = 0;i<(*d_m);i++)
	{
		d_r[n*(*d_m)+i] = d_a[n*(*d_m)+i] + d_b[n*(*d_m)+i];
	}  
}

int main(void)
{
int *a,*b,*r,m,n,i;
int *d_a,*d_b,*d_r,*d_m;

printf("Enter m,n : ");
scanf("%d %d",&m,&n);

a = (int*)malloc(m*n*sizeof(int));
b = (int*)malloc(m*n*sizeof(int));
r = (int*)malloc(m*n*sizeof(int));

printf("Enter matrix 1:\n");
for(i=0;i<m*n;i++)
{
	scanf("%d",&a[i]);	
}

printf("Enter matrix 2:\n");
for(i=0;i<m*n;i++)
{
	scanf("%d",&b[i]);	
}

hipMalloc((void **)&d_a,(m*n)*sizeof(int));
hipMalloc((void **)&d_b,(m*n)*sizeof(int));
hipMalloc((void **)&d_r,(m*n)*sizeof(int));
hipMalloc((void **)&d_m,sizeof(int));

hipMemcpy(d_a,a,(m*n)*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_b,b,(m*n)*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_r,r,(m*n)*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_m,&m,sizeof(int),hipMemcpyHostToDevice);


add<<<1,n>>>(d_a,d_b,d_r,d_m);
hipError_t error = hipGetLastError();
 if(error!= hipSuccess)
 {
	 	printf("%s\n",hipGetErrorString(error));
 }

hipMemcpy(r,d_r,(m*n)*sizeof(int),hipMemcpyDeviceToHost);
 
printf("Result matrix :\n");
for(i=0;i<m*n;i++)
{
	printf("%d ",r[i]);
	if((i+1)%m==0)
		printf("\n");	
}

}