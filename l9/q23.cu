#include<hip/hip_runtime.h>

#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void add(int* d_a,int* d_b,int* d_r)
{
	int col = threadIdx.x;
	int row = blockIdx.x;
	int size = blockDim.x;
	d_r[row*(size)+col] = d_a[row*(size)+col] + d_b[row*(size)+col];	  
}

int main(void)
{
int *a,*b,*r,m,n,i;
int *d_a,*d_b,*d_r;

printf("Enter m,n : ");
scanf("%d %d",&m,&n);

a = (int*)malloc(m*n*sizeof(int));
b = (int*)malloc(m*n*sizeof(int));
r = (int*)malloc(m*n*sizeof(int));

printf("Enter matrix 1:\n");
for(i=0;i<m*n;i++)
{
	scanf("%d",&a[i]);	
}

printf("Enter matrix 2:\n");
for(i=0;i<m*n;i++)
{
	scanf("%d",&b[i]);	
}

hipMalloc((void **)&d_a,(m*n)*sizeof(int));
hipMalloc((void **)&d_b,(m*n)*sizeof(int));
hipMalloc((void **)&d_r,(m*n)*sizeof(int));


hipMemcpy(d_a,a,(m*n)*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_b,b,(m*n)*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_r,r,(m*n)*sizeof(int),hipMemcpyHostToDevice);



add<<<m,n>>>(d_a,d_b,d_r);
hipError_t error = hipGetLastError();
 if(error!= hipSuccess)
 {
	 	printf("%s\n",hipGetErrorString(error));
 }

hipMemcpy(r,d_r,(m*n)*sizeof(int),hipMemcpyDeviceToHost);
 
printf("Result matrix :\n");
for(i=0;i<m*n;i++)
{
	printf("%d ",r[i]);
	if((i+1)%m==0)
		printf("\n");	
}

}