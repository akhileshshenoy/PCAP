#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

__global__ void add(float* x,float* y,int* n)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;

	if(id<*n)
		y[id]=sinf(x[id]);
}

int main()
{
	float a[100],res[100],*da,*db;
	int *dn;
	int n;

	printf("Enter size");
	scanf("%d",&n);
	printf("Enter the elements:");
	for(int i=0;i<n;i++)
		scanf("%f",&a[i]);

	hipMalloc((void**)&da,n*sizeof(float));
	hipMalloc((void**)&db,n*sizeof(float));
	hipMalloc((void**)&dn,sizeof(int));

	hipMemcpy(da,a,n*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dn,&n,sizeof(int),hipMemcpyHostToDevice);

	add<<<n,1>>>(da,db,dn);

	hipMemcpy(res,db,n*sizeof(float),hipMemcpyDeviceToHost);

	printf("\n N,1: ");

	for(int i=0;i<n;i++)
		printf("sine(%f)  =  %f\n",a[i],res[i]);

	hipFree(da);
	hipFree(db);
	hipFree(dn);
}