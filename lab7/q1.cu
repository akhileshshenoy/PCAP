#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void add(int* a,int* b,int* c,int* n)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;

	if(id<*n)
		c[id]=a[id]+b[id];
}

int main()
{
	int a[100],b[100],c[100],n,*da,*db,*dc;
	int *dn;

	printf("Enter size: ");
	scanf("%d",&n);
	printf("Enter elements for A: ");
	for(int i=0;i<n;i++)
		scanf("%d",&a[i]);
	printf("Enter elements for B: ");
	for(int i=0;i<n;i++)
		scanf("%d",&b[i]);

	hipMalloc((void**)&da,n*sizeof(int));
	hipMalloc((void**)&db,n*sizeof(int));
	hipMalloc((void**)&dc,n*sizeof(int));
	hipMalloc((void**)&dn,sizeof(int));

	hipMemcpy(da,a,n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(db,b,n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dn,&n,sizeof(int),hipMemcpyHostToDevice);

	add<<<n,1>>>(da,db,dc,dn);

	hipMemcpy(c,dc,n*sizeof(int),hipMemcpyDeviceToHost);

	printf("\nBlock size as N: ");

	for(int i=0;i<n;i++)
		printf("%d+%d=%d\n",a[i],b[i],c[i]);

	add<<<1,n>>>(da,db,dc,dn);

	hipMemcpy(c,dc,n*sizeof(int),hipMemcpyDeviceToHost);

	printf("\nN Threads: ");

	for(int i=0;i<n;i++)
		printf("%d+%d=%d\n",a[i],b[i],c[i]);

	int tsize=256;
	add<<<(n+tsize-1)/tsize,tsize>>>(da,db,dc,dn);

	hipMemcpy(c,dc,n*sizeof(int),hipMemcpyDeviceToHost);

	printf("\n256 threads: ");

	for(int i=0;i<n;i++)
		printf("%d+%d=%d\n",a[i],b[i],c[i]);

	hipFree(da);
	hipFree(db);
	hipFree(dc);
	hipFree(dn);
}