#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void q3(int* x,int* y,int* f,int* n,int* alpha)
{
	int id=blockIdx.x*blockDim.x+threadIdx.x;

	if(id<*n)
		f[id]=(*alpha)*x[id]+y[id];
}

int main()
{
	int a[100],b[100],c[100],n,alpha,*dalpha,*da,*db,*dc;
	int *dn;

	printf("Enter size: ");
	scanf("%d",&n);
	printf("Enter alpha: ");
	scanf("%d",&alpha);
	printf("Enter elements for A: ");
	for(int i=0;i<n;i++)
		scanf("%d",&a[i]);
	printf("Enter elements for B: ");
	for(int i=0;i<n;i++)
		scanf("%d",&b[i]);

	hipMalloc((void**)&da,n*sizeof(int));
	hipMalloc((void**)&db,n*sizeof(int));
	hipMalloc((void**)&dc,n*sizeof(int));
	hipMalloc((void**)&dn,sizeof(int));
	hipMalloc((void**)&dalpha,sizeof(int));

	hipMemcpy(da,a,n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(db,b,n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dn,&n,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dalpha,&alpha,sizeof(int),hipMemcpyHostToDevice);

	q3<<<n,1>>>(da,db,dc,dn,dalpha);

	hipMemcpy(c,dc,n*sizeof(int),hipMemcpyDeviceToHost);

	for(int i=0;i<n;i++)
		printf("%d*%d+%d=%d\n",alpha,a[i],b[i],c[i]);

	hipFree(da);
	hipFree(db);
	hipFree(dc);
	hipFree(dn);
	hipFree(dalpha);
}